#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define M 256  // Number of rows in A and C
#define K 512  // Number of columns in A and rows in B
#define N 256  // Number of columns in B and C
#define TILE_SIZE 16 //smaller tile size speeds things up
#define BLOCK_SIZE 16 //block size

__global__ void tileMatMul(float *A, float *B, float *C, int m, int k, int n)
{
    int by = blockIdx.y; //initialize indexes as variables for simplicity
    int bx = blockIdx.x;

    int ty = threadIdx.y;
    int tx = threadIdx.x;
    
    int row = by * blockDim.y + ty;
    int column = bx * blockDim.x + tx;

    __shared__ float shA[TILE_SIZE][TILE_SIZE]; //create shared memory for tile matrixes
    __shared__ float shB[TILE_SIZE][TILE_SIZE];

    float sum = 0.0f;
    
    for(int tile_num = 0; tile_num < ceil((float)k/TILE_SIZE); tile_num++) //main algo
    {
        if((row < m) && ((tile_num * TILE_SIZE + tx) < k)) //if thread doesnt load a value in shared mem
        {
            //map global index to shared index for matrix A
            shA[ty][tx] = A[(row * k) + (tile_num * TILE_SIZE + tx)]; 
        }
        else
        {
            shA[ty][tx] = 0.0f;
        }


        if(((tile_num * TILE_SIZE + ty) < k) && (column < n))
        {
            //map global index to shared index for matrix B
            shB[ty][tx] = B[((tile_num * TILE_SIZE + ty) * n) + column]; 
        }
        else
        {
            shB[ty][tx] = 0.0f;
        }
        __syncthreads();

        for(int k = 0; k < TILE_SIZE; k++)
        {
            sum += shA[ty][k] * shB[k][tx];
        }
        __syncthreads();
    }  
    if((row < m) && (column < n))
    {
        C[row * n + column] = sum;
    }
}

void gen_matrix(float *mat, int row, int column) //populates matrix with values
{
    for(int i = 0; i < row * column; i++)
    { 
        mat[i] = ((float)rand() / RAND_MAX);
         
    }
}

double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
}


int main()
{
    float *d_A, *d_B, *d_C; //initialize device matrices
    
    int size_A = M * K * sizeof(float); //calculate bytes needed for matrix A
    int size_B = K * N * sizeof(float); //calculate bytes needed for matrix B
    int size_C = M * N * sizeof(float); //calculate bytes needed for matrix C
  
    
    hipMalloc(&d_A, size_A);   //allocate space on device (GPU)
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    float *h_A = (float *)malloc(size_A); //allocate space on host (CPU)
    float *h_B = (float *)malloc(size_B);
    
    gen_matrix(h_A, M, K); //populates matrix A and B with random values
    gen_matrix(h_B, K, N);

    // Copy matrices from host to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    // Kernel launch configuration
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);

    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 3; i++) {
        tileMatMul<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, K, N);
        hipDeviceSynchronize();
    }

    printf("Benchmarking GPU implementation...\n");
    double gpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        tileMatMul<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, K, N);
        hipDeviceSynchronize();
        double end_time = get_time();
        gpu_total_time += end_time - start_time;
    }
    double gpu_avg_time = gpu_total_time / 20.0;

    // Print results
    printf("GPU average time: %f microseconds\n", (gpu_avg_time * 1e6f));

    // Free host memory
    free(h_A);
    free(h_B);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
